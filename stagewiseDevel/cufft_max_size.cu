
#include <iostream>
#include <string.h>
#include <math.h>
using namespace std;
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <float.h>
#include <fftw3.h>
#define NX 512
#define NY 512
#define NZ 1024
#define K 64  //dimension of small cube
#define BATCH 1
#define NRANK 3


// h_a is array reading from exported text of matlab
hipError_t PerfCuFFT(int argc, char **argv, hipfftDoubleComplex *h_a, hipfftDoubleComplex *result){
	hipError_t cudaStatus;
	hipfftHandle plan;
	hipfftDoubleComplex *data;
	hipfftDoubleComplex *d_a;
	int n[NRANK] = { NX, NY, NZ };
	int count;
	// Choosing CUDA device with newer architect
	//int dev = findCudaDevice(argc, (const char **)argv);


	hipMalloc((void**)&data, sizeof(hipfftDoubleComplex)*(NX*NY*NZ)*BATCH);
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		goto Error;
	}

	hipMalloc((void**)&d_a, sizeof(hipfftDoubleComplex)*NX*NY*NZ);
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_a, h_a, sizeof(hipfftDoubleComplex)*NX*NY*NZ, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	/* Create a 3D FFT plan. */
	if (hipfftPlanMany(&plan, NRANK, n,
		NULL, 1, NX*NY*NZ, // *inembed, istride, idist
		NULL, 1, NX*NY*NZ, // *onembed, ostride, odist
		HIPFFT_Z2Z, BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: Plan creation failed");
		goto Error;
	}

	/* Use the CUFFT plan to transform the signal in place. */
	if (hipfftExecZ2Z(plan, d_a, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT error: ExecZ2Z Forward failed");
		goto Error;
	}



	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, data, sizeof(hipfftDoubleComplex)*(NX*NY*NZ)*BATCH, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	//print result
	printf("CUFFT\n");
	count= 0;
	while(count<NX*NY*NZ){
		printf("%f + i %f\n",result[count].x, result[count].y);
		++count;}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipGetLastError();

Error:
	hipfftDestroy(plan);
	hipFree(data);
	hipFree(d_a);

	return cudaStatus;
}








int main(int argc, char **argv){
	hipfftDoubleComplex *data = new hipfftDoubleComplex[NX*NY*NZ];
	hipfftDoubleComplex *result;
	double  *cufft_output = new double[2*NX*NY*NZ];//fftw_input is real
  int count;

	result = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(NX*NY*NZ)*BATCH);
	data = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(NX*NY*NZ)*BATCH);

	//create small data cube inside larger data cube
	count = 0;
	for(int i=0;i<K;i++){
		for(int j=0;j<K;j++){
			for(int k=0;k<K;k++){
				data[NX*NY*i + NX*j + k ].x=i+j+k+0.3; //arbitrary value
				data[NX*NY*i + NX*j + k].y=0;
			}}}

			//pass same input to gpu. 'data' from host side.

			// Running cuFFT
			hipError_t cudaStatus = PerfCuFFT(argc, argv, data, result);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "CuFFT failed!");
				return 1;
			}


			cudaStatus = hipDeviceReset();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceReset failed!");
				return 1;
			}


	delete [] data;
	delete [] result;
	return 0;
}
