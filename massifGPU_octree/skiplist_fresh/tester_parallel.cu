#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <thrust/sort.h>
#include "skip_parallel.h"

__global__ void add(Skiplist *sl, OctreeNode a, int N)
{

    skiplist_insert(sl, a);
}

int main(void)
{
  OctreeNode *a = (OctreeNode*)malloc( sizeof(OctreeNode));
  OctreeNode *a_dev;
  int i,N;
  Skiplist *sl;
 

   N=1;
  // set heap size of 128 MB.
  CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024));
  size_t limit;
  hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
  printf("heap limit = %lu\n", limit);

  srand(time(NULL));
  CHECK(hipMalloc(&a_dev, sizeof(OctreeNode)));

  a->val = 4;
  printf("done initializing\n");

  sl = skiplist_create();
  CHECK(hipMemcpy(a_dev, a, sizeof(OctreeNode), hipMemcpyHostToDevice));
/*
  add<<<100, 320>>>(sl, a_dev, N);
  CHECK(hipDeviceSynchronize());
  printf("done inserting.\n");

  result = skiplist_gather(sl, &result_dim);
  result_sorted = (int *)malloc(result_dim * sizeof(int));
  memcpy(result_sorted, result, result_dim * sizeof(int));

  printf("done gathering.\n");
  printf("result_dim = %d\n", result_dim);

  thrust::sort(result_sorted, result_sorted + result_dim);
  printf("done sorting.\n");
  for (i = 0; i < result_dim; i++)
    if (result[i] != result_sorted[i])
      printf("mismatch at %d\n", i);
  printf("done checking.\n");

  free(a);
  free(result);
  free(result_sorted);
  hipFree(a_dev);
  printf("starting skiplist_destroy...\n");
  skiplist_destroy(sl);
*/
  return 0;
}
