#include <iostream>
#include <string.h>
#include <math.h>
#include <chrono>
using namespace std;
using namespace std::chrono;
#include "inputs.h"
#include "callbacks.cu"
#include "helperfunctions.h"
#include "fftwfunctions.h"
/*******************************

 Main function

*********************************/





int main(int argc, char **argv){

  //Host variables
	hipfftDoubleComplex *data;
	hipfftDoubleComplex *small_cube;
	hipfftDoubleComplex *result;
  
  unsigned long long int NPOINTS;
 
  NPOINTS= NX*NY*NZ;
  cout << NPOINTS <<endl;
  double *fftw_input;// = new double[NPOINTS];
  double *fftw_output;// = new double[2*NX*NY*NZ];
  double  *cufft_output;// = new double[2*NX*NY*NZ];
  int count;
  int correct;
  fftw_plan plan3d[2];
  fftw_plan plan3dinv[2];
  //Device variables
  hipfftDoubleComplex *d_result;
  hipfftDoubleComplex *d_a;
  int final_samples;
  final_samples =  (K*K + (NX*NY - K*K)/(DS*DS))*K + (NX*NY/(DS*DS))*(NZ-K)/DS;

  hipfftDoubleComplex* unsampled_result;

 	//allocating host side arrays
	result = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(final_samples));
  unsampled_result=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*NX*NY*((NZ-K)/DS));
	data = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(NX*NY*NZ));
  small_cube = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(K*K*K));
  fftw_input = (double*)malloc(sizeof(double)*2*NPOINTS);
  fftw_output = (double*)malloc(sizeof(double)*2*NPOINTS);
  cufft_output = (double*)malloc(sizeof(double)*2*NPOINTS);
  // Choosing CUDA device with newer architect
  //int dev = findCudaDevice(argc, (const char **)argv);

  //allocating device side arrays
  hipMalloc((void**)&d_a, sizeof(hipfftDoubleComplex)*K*K*K);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return 0;
  }


  //**TEMPORARY** the output is going to materialize the full cube for simplicity
  hipMalloc((void**)&d_result, sizeof(hipfftDoubleComplex)*final_samples);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return 0;
  }




cout<<"creating data"<<endl;
//create small data cube inside larger data cube
count = 0;
for(int i=0;i<K;i++){
	for(int j=0;j<K;j++){
	  for(int k=0;k<K;k++){
	    data[NX*NY*i + NX*j + k ].x= i*j*k+0.3 ;//arbitrary value
	    data[NX*NY*i + NX*j + k].y=0;

	    small_cube[K*K*i + K*j + k].x = i*j*k + 0.3; //same value as data
	    small_cube[K*K*i + K*j + k].y=0;

	  }}}

/*

for(int i=0;i<K;i++){
  for(int j=0;j<K;j++){
    for(int k=0;k<K;k++){
      cout<< "data " << data[NX*NY*i + NX*j + k ].x << endl;

    }}}

    for(int i=0;i<K;i++){
      for(int j=0;j<K;j++){
        for(int k=0;k<K;k++){

          cout<< "small cube " <<small_cube[K*K*i + K*j + k].x << endl;


        }}}
*/


  // Running cuFFT

  cout << "Run cufft" <<endl;
	auto start = high_resolution_clock::now();
  hipError_t cudaStatus = minibatch_CuFFT(argc, argv, small_cube, result, d_a, d_result, unsampled_result);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "CuFFT failed!");
    return 1;
  }
	auto stop = high_resolution_clock::now();
	auto durationCUDA = duration_cast<microseconds>(stop - start);

/*
  cout<< "copy result into double array"<< endl;
  //put result in cufft_output
  count = 0;
  for(int i=0;i<NZ;i++){
  for(int j=0;j<NY;j++){
  for(int k=0;k<NX;k++){
  cufft_output[count]= result[NX*NY*i + NX*j + k ].x;
  cufft_output[count+1] = result[NX*NY*i + NX*j + k].y;
  count = count + 2;
}}}

*/


cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceReset failed!");
  return 1;
}
/******************************
*
*
*  Create FFTW plan on CPU
*  and compute it for comparison
*******************************/

printf("creating fftw plan\n");


start = high_resolution_clock::now();
create_3Dplan_forward(plan3d, fftw_input, fftw_output, NX, NY, NZ);
create_3Dplan_inverse(plan3dinv, fftw_input, fftw_output, NX, NY, NZ);


//input for fftw on cpu..convert data from double to fftw_complex format
cout<<"fftw input"<<endl;
count = 0;
for(int i=0;i<NZ;i++){
	for(int j=0;j<NY;j++){
		for(int k=0;k<NX;k++){
			fftw_input[count]= data[NX*NY*i + NX*j + k].x;
			fftw_input[count+1] = data[NX*NY*i + NX*j + k].y;
			count=count+2;
		}}}
cout<<"end of input, count="<< count << endl;

//execute fftw
cout<<"executing FFTW forward plan"<<endl;

execute_fftw_3d_plan_forward(plan3d, fftw_input, fftw_output);

//multiply by green's
count = 0;
for(int i=0;i<NZ;i++){
	for(int j=0;j<NY;j++){
		for(int k=0;k<NX;k++){
			fftw_input[count]= fftw_output[count]*2.0;
			fftw_input[count+1] = 0.0;
			count=count+2;
		}}}

cout<<"executing FFTW plan and printing output"<<endl;
execute_fftw_3d_plan_inverse(plan3dinv, fftw_input, fftw_output);


stop = high_resolution_clock::now();
auto durationFFTW = duration_cast<microseconds>(stop - start);



 if(TO_PRINT==1){


		 printResult(result, final_samples);

		 cout<< "CUFFT unsampled first plane"<<endl;
		 count = 0;
		 while(count<NX*NY){
			 cout<< count << ": CUFFT :" << unsampled_result[count].x <<"," << unsampled_result[count].y << endl;
			 count = count + 1;
		 }
		 cout<< "FFTW first plane"<<endl;
		 count = 0;
		 while(count<NX*NY){
			 cout<< count << ": FFTW:" << fftw_output[2*count] <<"," << fftw_output[2*count+1] << endl;
			 count = count + 1;
		 }

}
else{
 //output is too large, only print few values

		 cout<< "First few values of CUFFT output"<<endl;
		 count = 0;
		 while(count<20){
			 cout<< count << ": CUFFT:" << unsampled_result[count].x <<"," << unsampled_result[count].y << endl;
			 count = count + 1;
		 }
		 cout<< "First few values of FFTW output"<<endl;
		 count = 0;
		 while(count<20){
			 cout<< count << ": FFTW:" << fftw_output[2*count] <<"," << fftw_output[2*count+1] << endl;
			 count = count + 1;
		 }


}


//Print timing info
cout << "CUDA time duration (plan create + execute):" << double(durationCUDA.count())/1000000 << endl ;
cout << "FFTW time duration (plan create + execute):" << double(durationFFTW.count())/1000000 << endl ;

fftw_destroy_plan(*plan3d);
fftw_destroy_plan(*plan3dinv);


delete [] data;
delete [] result;
delete [] fftw_input;
delete [] fftw_output;
delete [] small_cube;
return 0;
}
