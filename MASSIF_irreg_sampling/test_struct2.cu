
#include <hip/hip_runtime.h>
#include <stdio.h>

class CudaInput
{
public:
int* octree;
double* result;

CudaInput(int final_samples) {
    octree = new int[10];

    result=  new double[final_samples];

    result[1]=10.0;
}
};

__global__ void useClass(CudaInput *cudaClass)
{   printf("i want to print\n" );
    printf("%lf\n", cudaClass->result[1]);
};




int main()
{
    CudaInput c(5);
    // create class storage on device and copy top level class
    CudaInput *d_c;

    hipMalloc((void **)&d_c, sizeof(CudaInput));
    hipMemcpy(d_c, &c, sizeof(CudaInput), hipMemcpyHostToDevice);
    // make an allocated region on device for use by pointer in class
    int *temp_octree;
    double *temp_result;


    hipMalloc((void **)&temp_octree, sizeof(int)*10);
    hipMemcpy(temp_octree, c.octree, sizeof(int)*10, hipMemcpyHostToDevice);

    hipMalloc((void **)&temp_result, sizeof(double)*5);
    hipMemcpy(temp_result, c.result, sizeof(double)*5, hipMemcpyHostToDevice);
    // copy pointer to allocated device storage to device class
    hipMemcpy(&(d_c->octree), &temp_octree, sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(&(d_c->result), &temp_result, sizeof(double *), hipMemcpyHostToDevice);
    useClass<<<1,1>>>(d_c);
    hipDeviceSynchronize();
    return 0;
}
