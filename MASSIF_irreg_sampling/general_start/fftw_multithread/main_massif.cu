#include <iostream>
#include <string.h>
#include <math.h>
#include <chrono>
using namespace std;
using namespace std::chrono;
#include "inputs.h"
#include "callbacks.cu"
#include "helperfunctions.h"
#include "fftwfunctions.h"
#include "octree_table_host.cu"
#include <fftw3.h>
//#include <fftw_threads.h>
/*******************************

 Main function

*********************************/





int main(int argc, char **argv){

  //Host variables

	hipfftDoubleComplex *data;
	hipfftDoubleComplex *small_cube;
	hipfftDoubleComplex *result1;
	hipfftDoubleComplex* unsampled_result;
  double *fftw_input = new double[2*NX*NY*NZ];
  double *fftw_output = new double[2*NX*NY*NZ];
	double  *cufft_output = new double[2*NX*NY*NZ];
  int count;

  fftw_plan plan3d[2];
  fftw_plan plan3dinv[2];

  //Device variables
  hipfftDoubleComplex *d_result;
  hipfftDoubleComplex *d_a;
  int final_samples;
	int blocks;
	int *octreeTable;
	int XB, YB, ZB;
	int *ds_rates;
	XB = NX/OCTREE_FINEST;
	YB = NY/OCTREE_FINEST;
	ZB = NZ/OCTREE_FINEST;
	blocks = XB*YB*ZB;

	//init fftw threads
	int fftw_status;

	fftw_init_threads();
	fftw_plan_with_nthreads(NTHREADS);
	//if (fftw_status!=0){
	//	printf("Code = %d, Error in FFTW threads\n",fftw_status );
	//	return 0;
	//}



  ds_rates= (int*)malloc(sizeof(int)*blocks);
	octreeTable = (int*)malloc(sizeof(int)*blocks*5);

  final_samples = octree_table_construct(ds_rates, octreeTable);


	//allocating device side arrays
	hipMalloc((void**)&d_a, sizeof(hipfftDoubleComplex)*K*K*K);
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return 0;
	}


	//Store the output samples in this array
	hipMalloc((void**)&d_result, sizeof(hipfftDoubleComplex)*final_samples);
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return 0;
	}



 	//allocating host side arrays. temporary use of uniform DS rate
	result1 = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(final_samples));
  unsampled_result=(hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*NX*NY*((NZ-K)/DS));
	data = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(NX*NY*NZ));
  small_cube = (hipfftDoubleComplex*)malloc(sizeof(hipfftDoubleComplex)*(K*K*K));


  // Choosing CUDA device with newer architect
  //int dev = findCudaDevice(argc, (const char **)argv);





cout<<"creating data"<<endl;
//create small data cube inside larger data cube
count = 0;
for(int i=startZ;i<startZ + K;i++){
	for(int j=startY;j< startY + K;j++){
	  for(int k=startX;k< startX+K;k++){
	    data[NX*NY*i + NX*j + k ].x= 100 + i + j + k ;
	    data[NX*NY*i + NX*j + k].y=0;

	    small_cube[K*K*(i-startZ) + K*(j-startY) + (k-startX)].x = 100 + i + j + k ; //same value as data
	    small_cube[K*K*(i-startZ) + K*(j-startY) + (k-startX)].y=0;

	  }}}




  // Running cuFFT

  cout << "Run cufft" <<endl;
	auto start = high_resolution_clock::now();
  hipError_t cudaStatus = minibatch_CuFFT(argc, argv, small_cube, result1, d_a, d_result, unsampled_result, &final_samples);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "CuFFT failed!");
    return 1;
  }
	auto stop = high_resolution_clock::now();
	auto durationCUDA = duration_cast<microseconds>(stop - start);

/*
  cout<< "copy result into double array"<< endl;
  //put result in cufft_output
  count = 0;
  for(int i=0;i<NZ;i++){
  for(int j=0;j<NY;j++){
  for(int k=0;k<NX;k++){
  cufft_output[count]= result[NX*NY*i + NX*j + k ].x;
  cufft_output[count+1] = result[NX*NY*i + NX*j + k].y;
  count = count + 2;
}}}

*/




cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
  fprintf(stderr, "hipDeviceReset failed!");
  return 1;
}
/******************************
*
*
*  Create FFTW plan on CPU
*  and compute it for comparison
*******************************/



printf("creating fftw plan\n");


start = high_resolution_clock::now();
create_3Dplan_forward(plan3d, fftw_input, fftw_output, NX, NY, NZ);
create_3Dplan_inverse(plan3dinv, fftw_input, fftw_output, NX, NY, NZ);


//input for fftw on cpu..convert data from double to fftw_complex format
cout<<"fftw input"<<endl;
count = 0;
for(int i=0;i<NZ;i++){
	for(int j=0;j<NY;j++){
		for(int k=0;k<NX;k++){

			if((i>=startZ)&&(i<startZ + K) && (j>=startY)&&(j<startY + K) && (k>=startX)&&(k<startX + K) ){
				fftw_input[count]= data[NX*NY*i + NX*j + k].x;
				fftw_input[count+1] = data[NX*NY*i + NX*j + k].y;

				//cout<< fftw_input[count] << endl ;
		}
		else{
				fftw_input[count]= 0.0;
				fftw_input[count+1] = 0.0;

		}

			count=count+2;
		}}}
cout<<"end of input, count="<< count << endl;

//execute fftw
cout<<"executing FFTW forward plan"<<endl;

execute_fftw_3d_plan_forward(plan3d, fftw_input, fftw_output);

//multiply by green's
count = 0;
for(int i=0;i<NZ;i++){
	for(int j=0;j<NY;j++){
		for(int k=0;k<NX;k++){
			fftw_input[count]= fftw_output[count]*2.0;
			fftw_input[count+1] = 0.0;
			count=count+2;
		}}}

cout<<"executing FFTW plan and printing output"<<endl;
execute_fftw_3d_plan_inverse(plan3dinv, fftw_input, fftw_output);

write_fftw_to_csv(fftw_output);

stop = high_resolution_clock::now();
auto durationFFTW = duration_cast<microseconds>(stop - start);



 if(TO_PRINT==1){


		// printResult(result, final_samples);

		 cout<< "CUFFT unsampled first plane"<<endl;
		 count = 7*NX*NY;
		 while(count<8*NX*NY){
			 cout<< count << ": CUFFT :" << unsampled_result[count].x <<"," << unsampled_result[count].y << endl;
			 count = count + 1;
		 }
		 cout<< "FFTW first plane"<<endl;
		 count =7*NX*NY;
		 while(count<8*NX*NY){
			 cout<< count << ": FFTW:" << fftw_output[2*count] <<"," << fftw_output[2*count+1] << endl;
			 count = count + 1;
		 }

}
else{
 //output is too large, only print few values

		 cout<< "First few values of CUFFT output"<<endl;
		 count = 0;
		 while(count<5){
			 cout<< count << ": CUFFT:" << unsampled_result[count].x <<"," << unsampled_result[count].y << endl;
			 count = count + 1;
		 }
		 cout<< "First few values of FFTW output"<<endl;
		 count = 0;
		 while(count<5){
			 cout<< count << ": FFTW:" << fftw_output[2*count] <<"," << fftw_output[2*count+1] << endl;
			 count = count + 1;
		 }


}

/*
//sum of squares of fftw
count = 0;
double sum = 0;
while(count<NX*NY*NZ){
	sum = sum +  fftw_output[2*count]*fftw_output[2*count] +  fftw_output[2*count+1] *fftw_output[2*count+1] ;
	count = count + 1;
}

cout << "SUM OF SQUARES OF FFTW = " << sum << endl;
*/
//Print timing info
cout << "CUDA time duration (plan create + execute):" << double(durationCUDA.count())/1000000 << endl ;
cout << "FFTW time duration (plan create + execute):" << double(durationFFTW.count())/1000000 << endl ;


fftw_destroy_plan(*plan3d);
fftw_destroy_plan(*plan3dinv);


delete [] data;
delete [] result1;
delete [] fftw_input;
delete [] fftw_output;
delete [] small_cube;

return 0;

}
